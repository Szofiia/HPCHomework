#include "hip/hip_runtime.h"
﻿#include "hip/hip_runtime.h"
#include ""
#include <hip/hip_runtime.h>
#include <hiprand.h>
#include <hiprand/hiprand_kernel.h>

#include <iostream>

#include "Watermarking_CUDA.h"

// Numer of threads

// Cuda error handling
#define CUDA_CALL(x) do { if((x) != hipSuccess) { \
    printf("Error at %s:%d -- %s\n",__FILE__,__LINE__, hipGetErrorString(x)); \
    return;}} while(0)

#define CURAND_CALL(x) do { if((x)!=HIPRAND_STATUS_SUCCESS) { \
    printf("Error at %s:%d\n",__FILE__,__LINE__);\
    return;}} while(0)

// Calculates hiprandState.
//
// state - state to be generated
// seed - unified state
// N - number of random states generated
__global__ void setup_kernel_for_random(hiprandState* state, unsigned long seed, int N)
{
    int idx = blockIdx.x * blockDim.x + threadIdx.x;
    if(idx < N)
    hiprand_init(seed, idx, 0, &state[idx]);
}

// Calculates random values in the device
//
// global_state - global state used for generation
// random values - array containing the random values
__global__ void generate_for_random(hiprandState* global_state, float* random_values)
{
    int idx = blockIdx.x * blockDim.x + threadIdx.x;
    hiprandState localState = global_state[idx];
    float random = hiprand_uniform(&localState);
    random_values[idx] = random;
    global_state[idx] = localState;
}

// Calculates random values on the device and copies to the host,
// using hiprand Host API. (curandGenerator)
//
// host_data - copies random values to this container
// N - number of random values generated
void CalcRandWithHostAPI(float* host_data, int N)
{
    float *dev_data;
    hiprandGenerator_t generator;

    CUDA_CALL(hipMalloc((void**)&dev_data, N * sizeof(float)));

    CURAND_CALL(hiprandCreateGenerator(&generator, HIPRAND_RNG_PSEUDO_DEFAULT));
    CURAND_CALL(hiprandSetPseudoRandomGeneratorSeed(generator, 1234ULL));
    CURAND_CALL(hiprandGenerateUniform(generator, dev_data, N));

    CUDA_CALL(hipDeviceSynchronize());
    CUDA_CALL(hipMemcpy(host_data, dev_data, N * sizeof(*host_data), hipMemcpyDeviceToHost));

    // Cleanup
    CURAND_CALL(hiprandDestroyGenerator(generator));
    CUDA_CALL(hipFree(dev_data)); 
}

// Calculates random values on the device and copies to the host,
// using hiprand Device API. (hiprandState)
//
// host_data - copies random values to this container
// N - number of random values generated
void CalcRandWithDevAPI(float* host_data, int N)
{
    // 2D Thread blocks
    dim3 threads;
    if (N > 1024)
    {
        threads = dim3(512, 1);
    }
    else
    {
        threads = dim3(N, 1);
    }

    int blocks_count = floor(N / threads.x);
    dim3 blocks = dim3(blocks_count, 1);

    hiprandState* dev_states;
    float* dev_random_values;

    CUDA_CALL(hipMalloc(&dev_states, N * sizeof(hiprandState)));
    CUDA_CALL(hipMalloc(&dev_random_values, N * sizeof(*host_data)));

    // Setting up the random state
    setup_kernel_for_random << <blocks, threads >> > (dev_states, time(NULL), N);

    // Generating random numbers
    generate_for_random << <blocks, threads >> > (dev_states, dev_random_values);

    CUDA_CALL(hipMemcpy(host_data, dev_random_values, N * sizeof(*host_data), hipMemcpyDeviceToHost));

    // Cleanup
    CUDA_CALL(hipFree(dev_states));
    CUDA_CALL(hipFree(dev_random_values));
}